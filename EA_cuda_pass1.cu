#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <math.h>
#include "cuPrintf.cuh"
#include "cuPrintf.cu"
#include <stdlib.h>
#include<float.h>
#include <stdio.h>
#include <hiprand/hiprand_kernel.h>

// Run with 
// nvcc EA_cuda_pass1.cu
// ./a.out

#define CUDA_SAFE_CALL(ans) { gpuAssert((ans), (char *)__FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, char *file, int line, bool abort=true)
{
  if (code != hipSuccess)
  {
    fprintf(stderr, "CUDA_SAFE_CALL: %s %s %d\n",
                                       hipGetErrorString(code), file, line);
    if (abort) exit(code);
  }
}

#define numberCities 15
#define PRINT_TIME   1
#define xLimit 5
#define yLimit 0
#define blocks 1
#define threads 10
#define population blocks*threads
#define iters 10000



__device__ float global_minDis = FLT_MAX;

#define IMUL(a, b) __mul24(a, b)

double interval(struct timespec start, struct timespec end)
{
  struct timespec temp;
  temp.tv_sec = end.tv_sec - start.tv_sec;
  temp.tv_nsec = end.tv_nsec - start.tv_nsec;
  if (temp.tv_nsec < 0) {
    temp.tv_sec = temp.tv_sec - 1;
    temp.tv_nsec = temp.tv_nsec + 1000000000;
  }
  return (((double)temp.tv_sec) + ((double)temp.tv_nsec)*1.0e-9);
}

void setup(float* x, float* y, int*in)
{
    int i;
    for(i=0; i<numberCities; i++)
    {
        x[i]= ((float)rand()/(float)RAND_MAX)*xLimit;
        y[i]= ((float)rand()/(float)RAND_MAX)*yLimit;
        in[i]=i;
    }

}

void swap(int* arr, int one, int two){
    int temp = arr[one];
    arr[one]= arr[two];
    arr[two]= temp;

}

void shuffle(int* random){
    int i, randN;
    

    for(i =0;i<numberCities;i++){
        randN=round(((float)rand()/(float)RAND_MAX)*(numberCities-1-i)+i);
        swap(random, i, randN);
    }
   
}

void assignGene(int* genes, float* x, float*y, int* ind){
    int i,j;
    int random[numberCities];

    for(i=0; i<numberCities; i++){
        random[i]= ind[i];
    }

    for(i =0; i<population; i++){
        shuffle(random);
        for(j=0; j<numberCities; j++){
            genes[i*numberCities+j]= random[j];
        }
    }
}

void printCityLoc (float *x, float* y){
    int i;

    for(i=0; i<numberCities; i++){
        printf("Index %d: x = %f y=%f \n", i, x[i], y[i]);
    }
}

__device__ float global_min = FLT_MAX;

__device__ float dist(float x1, float x2, float y1, float y2){
    float temp= ((x1-x2)*(x1-x2)) + ((y1-y2)*(y1-y2));
    temp = sqrt(temp);
    return temp;
}

__device__ void copyBest(int ind, int* genes, int* d_bestSoFar){

  int i;
  for(i=0; i<numberCities; i++){
    d_bestSoFar[i]= genes[ind*numberCities+i];
  }

}

__device__ void distanceCal(float* x, float* y, int* genes, float* dis, int* local_min_ind, float* local_min_dis, int* d_bestSoFar){

  int id= threadIdx.x;
  int j;
  int ind;
  float temp;
  __shared__ float sum[population];
  float minDis;

  sum[id]=0;
  for(j=0; j<numberCities-1; j++){
    temp = dist(x[genes[id*numberCities+j]], x[genes[id*numberCities+j+1]],y[genes[id*numberCities+j]], y[genes[id*numberCities+j+1]]);
    sum[id] = sum[id]+temp;
  }

  dis[id]= sum[id];
  __syncthreads();

if(id==0){
  minDis= sum[0];
  ind=0;
  for(j=1; j<population; j++)
  {
    if(sum[j]<minDis){
      minDis = sum[j];
      ind=j;
    }
  }

  if(minDis<global_min){
    global_min = minDis;
    copyBest(ind, genes, d_bestSoFar);
  }

  *local_min_ind= ind;
  *local_min_dis= minDis;

  
}

}

__device__ void printDist(float* dis){
  int id= threadIdx.x;
  printf("Ind #:%d, dist =%f\n", id, dis[id]);
}

__device__ void fitnessFun(float* fitVal, float* dis){
  int id= threadIdx.x;  
  float total;
  int i;
  if(id==0){
    total=0;
    for(i=0; i<population; i++){
      total = total+ dis[i];
    }
    for(i=0; i<population; i++){
      fitVal[i] = dis[i]/total ;
    }
    // printf("%f \n", total);
  }
  __syncthreads();

}

__device__ void mutate(int* genes, int* local_min_ind, float rate, hiprandState *state){

//SECOND TRY
  int num_chosen = int(rate*numberCities);
  int id= threadIdx.x;
  int temporary[numberCities];
  int i,j,hold;
  int flag;
  int ind = *local_min_ind;
  int randN= floor(hiprand_uniform(state+id)*(numberCities-1)); //should be changed to be random
  for(i=0; i<num_chosen; i++){
   temporary[i]= genes[ind*numberCities+ randN];
   randN++;
   randN = randN % numberCities;
 }

 int next =num_chosen;

 for(i=0; i<numberCities; i++){
   hold = genes[id*numberCities+i];
   flag =0;

   for(j=0; j<num_chosen; j++){
      if(hold == temporary[j]){
        flag =1;
        break;
      }
    }
    
    if (flag==0){
      temporary[next]=hold;
      next++;
    }

    if(next== numberCities){
      break;
    }
 }

 for(i=0; i<numberCities;i++){

   genes[id*numberCities+i] = temporary[i];
 }


}
//maybe add mutate here
__device__ void EA( int* genes, float* fitVal,int* local_min_ind, int iter, hiprandState *state){
  int id = threadIdx.x;
  float frac = iter/iters;
  float rate = hiprand_uniform(state+id) *(1-frac); //should be random
  mutate(genes, local_min_ind, rate, state);
}

__device__ void printRelevantInfo(float* local_min_dis){
    printf("Minimum Local Dis in this Iteration is %f. \nMinimum Global Dis is %f.\n\n\n", *local_min_dis, global_min);

}

__device__ void printGenes(int* genes){

  int i;
  int j;
  for(i=0; i<population; i++){
    for(j=0; j<numberCities; j++){
      printf("%d  ", genes[i*numberCities+j]);
    }
    printf("\n");
  }

}

__global__ void main_Kernel(float* x, float* y, int* genes, float* dis, int* local_min_ind, float* local_min_dis, int* d_bestSoFar, float* fitVal, hiprandState* state){

  int id= threadIdx.x;
  int iter =2;
  hiprand_init(1234+id, id, 0, &state[id]);

  distanceCal( x,  y,  genes,  dis, local_min_ind, local_min_dis, d_bestSoFar);
  if(id==0){
   printf("Iteration 1\n\n"); 
    printRelevantInfo(local_min_dis); 
  }
  
  while(iter<iters){
    fitnessFun(fitVal, dis);
    EA(genes, fitVal, local_min_ind, iter, state);
    distanceCal( x,  y,  genes,  dis, local_min_ind, local_min_dis, d_bestSoFar);

    if(id==0){
      printf("Iteration %d\n\n", iter);
      printRelevantInfo(local_min_dis);
    }
    iter++;
  }
}


int main(){
  srand(time(NULL));
  int i;

  hiprandState *d_state;
  hipMalloc((void**)&d_state, sizeof(hiprandState));

  // GPU Timing variables
    hipEvent_t start, stop;
    float elapsed_gpu;

  // Select GPU
  CUDA_SAFE_CALL(hipSetDevice(0));

 // Randomly chooses points that represents cities
  float h_xCoord[numberCities];
  float h_yCoord[numberCities];
  int indices[numberCities]; 
  setup(h_xCoord, h_yCoord, indices);

// Host Variables
  int* h_bestSoFar;
  //genes where each row represent one gene 
  int* h_genes;


// Device Variables
  float* d_xCoord;
  float* d_yCoord;
  float* d_dis;
  float* d_fitVal;
  int* d_local_popMinIndex;
  float* d_local_min_dist;
  int* d_bestSoFar;
  //genes where each row represent one gene 
  int* d_genes;


  // Size allocation for the genes array
  size_t allocSize2d = numberCities* sizeof(int)*population;
  CUDA_SAFE_CALL(hipMalloc((void**)&d_genes, allocSize2d));
  h_genes = (int*) malloc(allocSize2d);
  
  
  assignGene(h_genes, h_xCoord, h_yCoord, indices);

  // size allocation for minimum local index and distance
  size_t allocSize_fl =  sizeof(float);
  size_t allocSize_int =  sizeof(int);
  CUDA_SAFE_CALL(hipMalloc((void**)&d_local_min_dist, allocSize_fl));
  CUDA_SAFE_CALL(hipMalloc((void**)&d_local_popMinIndex, allocSize_int));

  // Size allocation for the x and y coordinates
  size_t allocSize_cities_fl = numberCities* sizeof(float);
  CUDA_SAFE_CALL(hipMalloc((void**)&d_xCoord, allocSize_cities_fl));
  CUDA_SAFE_CALL(hipMalloc((void**)&d_yCoord, allocSize_cities_fl));

  // Size allocation for the array of best sequence of indices
  size_t allocSize_cities_int = numberCities* sizeof(int);
  CUDA_SAFE_CALL(hipMalloc((void**)&d_bestSoFar, allocSize_cities_int));
  h_bestSoFar = (int*) malloc(allocSize_cities_int);

  
  size_t allocSize_pop_fl = population* sizeof(float);
  CUDA_SAFE_CALL(hipMalloc((void**)&d_dis, allocSize_pop_fl));
  CUDA_SAFE_CALL(hipMalloc((void**)&d_fitVal, allocSize_pop_fl));

  #if PRINT_TIME
    // Create the cuda events
    hipEventCreate(&start);
    hipEventCreate(&stop);
    // Record event on the default stream
    hipEventRecord(start, 0);
   #endif

  CUDA_SAFE_CALL(hipMemcpy(d_genes, h_genes,allocSize2d, hipMemcpyHostToDevice));
  CUDA_SAFE_CALL(hipMemcpy(d_xCoord, h_xCoord,allocSize_cities_fl, hipMemcpyHostToDevice));
  CUDA_SAFE_CALL(hipMemcpy(d_yCoord, h_yCoord,allocSize_cities_fl, hipMemcpyHostToDevice));
  CUDA_SAFE_CALL(hipMemcpy(d_bestSoFar, h_bestSoFar,allocSize_cities_int, hipMemcpyHostToDevice));
  

  main_Kernel <<<blocks, threads>>> (d_xCoord, d_yCoord, d_genes,d_dis, d_local_popMinIndex, d_local_min_dist, d_bestSoFar, d_fitVal, d_state);

  CUDA_SAFE_CALL(hipMemcpy(h_bestSoFar, d_bestSoFar,allocSize_cities_int, hipMemcpyDeviceToHost));  

  #if PRINT_TIME
    // Stop and destroy the timer
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed_gpu, start, stop);
    printf("\nTotal time on GPU is: %f(msec)\n", elapsed_gpu);
    hipEventDestroy(start);
    hipEventDestroy(stop);
  #endif
  
  for(i=0; i<numberCities; i++){
        printf("%d ", h_bestSoFar[i]);
  }

  printf("\n");

  
  // printCityLoc(h_xCoord, h_yCoord);


  CUDA_SAFE_CALL(hipFree(d_xCoord)); 
  CUDA_SAFE_CALL(hipFree(d_yCoord)); 
  CUDA_SAFE_CALL(hipFree(d_genes));


 free(h_genes);

    


  return 0;
}